#include "hip/hip_runtime.h"

// Based on CUDA SDK template from NVIDIA

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <assert.h>
#include <float.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_image.h>

#define max(a,b) (((a)>(b))?(a):(b))
#define min(a,b) (((a)<(b))?(a):(b))

#define MAX_BRIGHTNESS 255
 
// pixel base type
// Use int instead `unsigned char' so that we can
// store negative values.
typedef int pixel_t;


// harris detector code to run on the host
void harrisDetectorHost(const pixel_t *h_idata, const int w, const int h, 
                const int ws,               // window size
                const int threshold,        // threshold value to detect corners
                pixel_t * reference)
{
    int i,j,k,l;  // indexes in image
    int Ix, Iy;   // gradient in XX and YY
    int R;        // R metric
    int sumIx2, sumIy2, sumIxIy;

    for(i=0; i<h; i++) //height image
    {
        for(j=0; j<w; j++) //width image
        {
            reference[i*w+j]=h_idata[i*w+j]/4; // to obtain a faded background image
        }
    }

    for(i=ws+1; i<h-ws-1; i++) //height image
    {
        for(j=ws+1; j<w-ws-1; j++) //width image
        {
           sumIx2=0;sumIy2=0;sumIxIy=0;
           for(k=-ws; k<=ws; k++) //height window
              {
                  for(l=-ws; l<=ws; l++) //width window
                  {
                        Ix = ((int)h_idata[(i+k-1)*w + j+l] - (int)h_idata[(i+k+1)*w + j+l])/32;         
                        Iy = ((int)h_idata[(i+k)*w + j+l-1] - (int)h_idata[(i+k)*w + j+l+1])/32;         
                        sumIx2 += Ix*Ix;
                        sumIy2 += Iy*Iy;
                        sumIxIy += Ix*Iy;
                  }
              }

              R = sumIx2*sumIy2-sumIxIy*sumIxIy-0.05*(sumIx2+sumIy2)*(sumIx2+sumIy2);
              if(R > threshold) {
                   reference[i*w+j]=MAX_BRIGHTNESS; 
              }
        }
    }
}   

__global__ void kernel_Harris(pixel_t *dev_idata, const int w, const int h, 
                  const int ws, const int threshold, pixel_t *dev_odata)		
{
    int Ix, Iy;   // gradient in XX and YY
    int R;        // R metric
    int sumIx2=0, sumIy2=0, sumIxIy=0;

    int j = (blockIdx.x * blockDim.x) + threadIdx.x;
    int i = (blockIdx.y * blockDim.y) + threadIdx.y;

    // fade image
    dev_odata[i*w+j]=dev_idata[i*w+j]/4;

    if((i>= ws + 1) && (i < h-ws-1) && (j >= ws + 1) && (j < w-ws-1)) // only if valid interior region
    {
        for(int k=-ws; k<=ws; k++) //height window
        {
            for(int l=-ws; l<=ws; l++) //width window
            {
                Ix = ((int)dev_idata[(i+k-1)*w + j+l] - (int)dev_idata[(i+k+1)*w + j+l])/32;         
                Iy = ((int)dev_idata[(i+k)*w + j+l-1] - (int)dev_idata[(i+k)*w + j+l+1])/32;          
                sumIx2 += Ix*Ix;
                sumIy2 += Iy*Iy;
                sumIxIy += Ix*Iy;
            }
        }
    

        R = sumIx2*sumIy2-sumIxIy*sumIxIy-0.05*(sumIx2+sumIy2)*(sumIx2+sumIy2);
        
        // is a corner
        if(R > threshold) 
                dev_odata[i*w+j]=MAX_BRIGHTNESS;

    }
}

// harris detector code to run on the GPU
void harrisDetectorDevice(const pixel_t *h_idata, const int w, const int h, 
                  const int ws, const int threshold, 
                  pixel_t * h_odata)
{
    //TODO
    pixel_t *dev_idata, *dev_odata;
    pixel_t data_size;

    // full size for an image
    data_size = w * h * sizeof(pixel_t);
    printf("data_size %d\n", data_size);

    // Max number of threads
    int numThreads_x = 32;
	int numThreads_y = 32;
    
    // Resize for exact scaling
    while((w % numThreads_x ) != 0)
        numThreads_x = numThreads_x - 1;
        
	while((h % numThreads_y ) != 0)
        numThreads_y = numThreads_y - 1;

    // Number of blocks with that number of threads
    int numBlocks_x = ceil(w/numThreads_x);
	int numBlocks_y = ceil(h/numThreads_y);

    // CUDA dimensions
    dim3 dimBlock(numThreads_x, numThreads_y); //  threadsPerBlock
	dim3 dimGrid(numBlocks_x, numBlocks_y); // numBlocks

    // print dimensions
    printf ("w = %d\n", w);
	printf ("h = %d\n", h);
	printf("dimBlock = %d x %d \n", dimBlock.x, dimBlock.y);
	printf("dimGrid = %d x %d \n", dimGrid.x, dimGrid.y); 

    // memory allocation
    hipMalloc((void **)&dev_idata, data_size);
    hipMalloc((void **)&dev_odata, data_size);

    // copy image to device (CPU->GPU)
    hipMemcpy(dev_idata, h_idata, data_size, hipMemcpyHostToDevice);

    // Run corner detetion on GPU
    kernel_Harris<<<dimGrid, dimBlock>>>(dev_idata, w, h, ws, threshold, dev_odata);

    // Copy result from device to host (GPU->CPU)
    hipMemcpy(h_odata, dev_odata, data_size, hipMemcpyDeviceToHost);

    // free allocated memory
    hipFree(dev_idata);
    hipFree(dev_odata);

}

// print command line format
void usage(char *command) 
{
    printf("Usage: %s [-h] [-d device] [-i inputfile] [-o outputfile] [-r referenceFile] [-w windowsize] [-t threshold]\n",command);
}

// main
int main( int argc, char** argv) 
{

    // default command line options
    int deviceId = 0;
    char *fileIn        = (char *)"chess.pgm",
         *fileOut       = (char *)"resultCuda.pgm",
         *referenceOut  = (char *)"referenceCuda.pgm";
    unsigned int ws = 1, threshold = 500;

    // parse command line arguments
    int opt;
    while( (opt = getopt(argc,argv,"d:i:o:r:w:t:h")) !=-1)
    {
        switch(opt)
        {

            case 'd':
                if(sscanf(optarg,"%d",&deviceId)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;

            case 'i':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }

                fileIn = strdup(optarg);
                break;
            case 'o':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                fileOut = strdup(optarg);
                break;
            case 'r':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                referenceOut = strdup(optarg);
                break;
            case 'w':
                if(strlen(optarg)==0 || sscanf(optarg,"%d",&ws)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 't':
                if(strlen(optarg)==0 || sscanf(optarg,"%d",&threshold)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 'h':
                usage(argv[0]);
                exit(0);
                break;

        }
    }

    // select cuda device
    checkCudaErrors( hipSetDevice( deviceId ) );
    
    // create events to measure host harris detector time and device harris detector time

    hipEvent_t startH, stopH, startD, stopD;
    checkCudaErrors( hipEventCreate(&startH) );
    checkCudaErrors( hipEventCreate(&stopH)  );
    checkCudaErrors( hipEventCreate(&startD) );
    checkCudaErrors( hipEventCreate(&stopD)  );



    // allocate host memory
    pixel_t * h_idata=NULL;
    unsigned int h,w;

    //load pgm
    if (sdkLoadPGM<pixel_t>(fileIn, &h_idata, &w, &h) != true) {
        printf("Failed to load image file: %s\n", fileIn);
        exit(1);
    }

    // allocate mem for the result on host side
    pixel_t * h_odata   = (pixel_t *) malloc( h*w*sizeof(pixel_t));
    pixel_t * reference = (pixel_t *) malloc( h*w*sizeof(pixel_t));
 
    // detect corners at host

    checkCudaErrors( hipEventRecord( startH, 0 ) );
    harrisDetectorHost(h_idata, w, h, ws, threshold, reference);   
    checkCudaErrors( hipEventRecord( stopH, 0 ) ); 
    checkCudaErrors( hipEventSynchronize( stopH ) );

    // detect corners at GPU
    checkCudaErrors( hipEventRecord( startD, 0 ) );
    harrisDetectorDevice(h_idata, w, h, ws, threshold, h_odata);   
    checkCudaErrors( hipEventRecord( stopD, 0 ) ); 
    checkCudaErrors( hipEventSynchronize( stopD ) );
    
    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    float timeH, timeD;
    checkCudaErrors( hipEventElapsedTime( &timeH, startH, stopH ) );
    printf( "Host processing time: %f (ms)\n", timeH);
    checkCudaErrors( hipEventElapsedTime( &timeD, startD, stopD ) );
    printf( "Device processing time: %f (ms)\n", timeD);

    // save output images
    if (sdkSavePGM<pixel_t>(referenceOut, reference, w, h) != true) {
        printf("Failed to save image file: %s\n", referenceOut);
        exit(1);
    }
    if (sdkSavePGM<pixel_t>(fileOut, h_odata, w, h) != true) {
        printf("Failed to save image file: %s\n", fileOut);
        exit(1);
    }

    // cleanup memory
    free( h_idata);
    free( h_odata);
    free( reference);

    checkCudaErrors( hipDeviceReset() );
}
