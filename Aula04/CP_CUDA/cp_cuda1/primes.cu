
#include <hip/hip_runtime.h>
#include  <stdio.h> 
#include  <time.h> 

#define  NROWS (1024) 
#define  NCOLS (8) 
#define  SIZE (NROWS*NCOLS) 


int compare(int *a1, int *a2);


//  Kernel definition, see also section 2.1 of NVIDIA CUDA Programming Guide 
__global__  void primes(int *A, int *B) 
{ 
    // TODO: determine id
    int id;

    id = (blockIdx.y * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;


    if(id < SIZE)
    {
        if (A[id] == 0 || A[id] == 1)
        {
            B[id] = 0;
        }
        for (int i = 2; i <= A[id] / 2 && !B[id]; ++i) {
            if (A[id] % i == 0) 
            {
                B[id] = 1;
            }
        }
    }
} 


int primesHost(int A) 
{ 
    int B = 0;
    if (A == 0 || A == 1)
    {
        B = 0;
    }
    for (int i = 2; i <= A / 2 && !B; ++i) {
        if (A % i == 0) 
        {
            B = 1;
        }
    }
    return B;
} 

int  main(void) 
{ 
    int A[SIZE], D[SIZE], H[SIZE];
    int *devPtrA; 
    int *devPtrD; 
    int memsize = SIZE * sizeof(int); 
    float devExecTime;

    hipSetDevice(0);   // Select GPU device (can be 0 to 1)

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Initialize arrays
    srand (time(NULL));
    for(int i=0; i < SIZE; i++) 
    {
        A[i]=rand() % 100;
    }

    printf("Starting HOST...\n");

    for(int i=0; i < NCOLS; i++)
    {
        for(int j=0; j < NROWS; j++)
        {
            int id = j + i * NROWS;
            H[id] = primesHost(A[id]); /// host result...
        }
    }

    // Allocate device memory for A, B and D arrays
    hipMalloc((void**)&devPtrA, memsize); 
    hipMalloc((void**)&devPtrD, memsize); 

    printf("Starting DEVICE...\n");
    hipEventRecord(start);

    // Copy data (data to process) from host to device (from CPU to GPU)
    hipMemcpy(devPtrA, A, memsize,  hipMemcpyHostToDevice);

    // __global__ functions are called:  Func <<< dim grid, dim block >>> (parameter); 
    dim3 dimBlock(2, 2);
    dim3 dimGrid(SIZE / dimBlock.x, SIZE / dimBlock.y);
    

    // Execute the Kernel 
    primes <<<dimGrid, dimBlock>>> (devPtrA, devPtrD); 

    // Copy data from device (results) back to host 
    hipMemcpy(D, devPtrD, memsize,  hipMemcpyDeviceToHost); 

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&devExecTime, start, stop); //Exec time = elapsed time
    

    // Show results
    printf("     A      B       D      H\n");
    for (int i=0; i < SIZE; i++) 
    {
        printf("%2d: %4d -> %5d [%5d]\n", i, A[i], D[i], H[i]); 
    }

    printf("\nOutput arrays (H/D) are %s\n", compare(D, H) == 1 ? "EQUAL" : "DIFFERENT");

    printf("\nDevice execution time [ms]: %7.4f\n", devExecTime);

    // Free device memory
    hipFree(devPtrA);
    hipFree(devPtrD); 
} 

int compare(int *a1, int *a2)
{
    int i, j, equal = 1;
    for(j=0; (j < NROWS) && equal; j++)
    {
        for(i=0; (i < NCOLS) && equal; i++)
        {
            int id = i + j * NCOLS;
            if(a1[id] != a2[id])
                equal = 0;
        }
    }
    return equal;
}


